#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
 
#define ARRAY_SIZE 256
#define NUM_BLOCKS  8
#define THREADS_PER_BLOCK 32

/* results 1 block */
/*
elapsed time (milliseconds): 1.781760
Effective Bandwidth (GB/s): 0.001149
*/

/* results multiblock */
/*
elapsed time (milliseconds): 1.654784
Effective Bandwidth (GB/s): 0.001238k
*/

__global__ void negate(int *d_a){
    int idx = threadIdx.x;
    d_a[idx] = -1 * d_a[idx];
}
 
__global__ void negate_multiblock(int *d_a){
    int idx = threadIdx.x + (blockIdx.x * blockDim.x);
    d_a[idx] = -1 * d_a[idx];
}
 
int main(int argc, char *argv[]){
    int *h_a, *h_out;
    int *d_a;
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int i;
    size_t siz_b = ARRAY_SIZE * sizeof(int);
 
    hipHostMalloc((int**)&h_a, siz_b, hipHostMallocDefault);
    hipHostMalloc((int**)&h_out, siz_b, hipHostMallocDefault);

    for (i = 0; i < ARRAY_SIZE; i++) {
        h_a[i] = i;
        h_out[i] = 0;
    }

    hipMalloc((void**)&d_a, siz_b);

    hipMemcpy(d_a, h_a, siz_b, hipMemcpyHostToDevice);

    //dim3 blocksPerGrid( ); 
    //dim3 threadsPerBlock( );
    hipEventRecord(start);
    negate_multiblock<<< NUM_BLOCKS , THREADS_PER_BLOCK >>>( d_a );
    hipEventRecord(stop);
    //negate_multiblock<<<,>>>();
    //cudaDeviceSynchronize();
 
    hipMemcpy(h_out, d_a, siz_b, hipMemcpyDeviceToHost);
 
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("elapsed time (milliseconds): %f\n",milliseconds);
    double bytes = ARRAY_SIZE * sizeof(int) * 2;  // Read + Write
    printf("Effective Bandwidth (GB/s): %f\n", bytes/milliseconds/1e6);

    printf("Results: ");
    for (i = 0; i < ARRAY_SIZE; i++) {
      printf("%d, ", h_out[i]);
    }
    printf("\n\n");
 
    hipHostFree(h_a);
    hipHostFree(h_out);
    hipFree(d_a);
 
    return 0;
}